#include<time.h>
#include<math.h>
#include<stdio.h>
#include<stdlib.h>
#include<assert.h>
#include<hip/hip_runtime.h>


#define NTPB 128            /* Number of Threads Per Block */


__device__ inline void myAtomicAdd(double *address, double value)  //See CUDA official forum
 {
    unsigned long long oldval, newval, readback;
 
    oldval = __double_as_longlong(*address);
    newval = __double_as_longlong(__longlong_as_double(oldval) + value);
    while ((readback=atomicCAS((unsigned long long *)address, oldval, newval)) != oldval)
    {
        oldval = readback;
        newval = __double_as_longlong(__longlong_as_double(oldval) + value);
    }
 }

__global__ void integrater(float *x, float *y, float *z, float *u, float *v, float *I1, int n){
    int i;
    int iglob = threadIdx.x + blockIdx.x*blockDim.x; 
    int iloc  = threadIdx.x                        ;
    extern __shared__ float block_cache[]; 

    if (iglob < n)
	block_cache[iloc] = expf(-x[iglob]*x[iglob] - y[iglob]*y[iglob]-z[iglob]*z[iglob]-v[iglob]*v[iglob]-u[iglob]*u[iglob]);/*main function eval*/
    else
	block_cache[iloc] = 0;

    __syncthreads();

    /* on the "master thread" of each block" sum the pairwise products
       on that block into the block's portion of the global sum */
    if (iloc == 0){
	float sum = 0.0;
	for (i=0;i<NTPB;++i)
	    sum += block_cache[i];
	atomicAdd(I1,sum);  
    }

}


int main(int argc, char **argv){
    float *x,   *y,   *z, *u, *v, *I1;       /* host pointers */
    float *x_d, *y_d, *z_d, *u_d, *v_d, *I1_d;     /* device pointers */
    int i,n;                  /* vector length */
    hipEvent_t start, stop;  /* timers */
    float times;
    float actual = .232322;

    n = atoi(argv[1]);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* allocate host memory */
    assert (hipHostMalloc((void **) &x, n*sizeof(float), hipHostMallocDefault) == hipSuccess);
    assert (hipHostMalloc((void **) &y, n*sizeof(float), hipHostMallocDefault) == hipSuccess);
    assert (hipHostMalloc((void **) &z, n*sizeof(float), hipHostMallocDefault) == hipSuccess);
    assert (hipHostMalloc((void **) &u, n*sizeof(float), hipHostMallocDefault) == hipSuccess);
    assert (hipHostMalloc((void **) &v, n*sizeof(float), hipHostMallocDefault) == hipSuccess);
    assert (hipHostMalloc((void **) &I1, 1*sizeof(float), hipHostMallocDefault) == hipSuccess);

    srand((time(NULL)));
    for (i=0;i<n;++i){
	x[i] = (float)rand()/(float)(RAND_MAX-1);
	y[i] = (float)rand()/(float)(RAND_MAX-1);
	z[i] = (float)rand()/(float)(RAND_MAX-1);
	u[i] = (float)rand()/(float)(RAND_MAX-1);
	v[i] = (float)rand()/(float)(RAND_MAX-1);
	
    }

    *I1 = 0.0;
    /* allocate memory on device */
    assert (hipMalloc((void **) &x_d, n*sizeof(float)) == hipSuccess);
    assert (hipMalloc((void **) &y_d, n*sizeof(float)) == hipSuccess);
    assert (hipMalloc((void **) &z_d, n*sizeof(float)) == hipSuccess);
    assert (hipMalloc((void **) &u_d, n*sizeof(float)) == hipSuccess);
    assert (hipMalloc((void **) &v_d, n*sizeof(float)) == hipSuccess);
    assert (hipMalloc((void **) &I1_d, 1*sizeof(float)) == hipSuccess);

    /* copy host data to device pointers */
    assert(hipMemcpy(x_d,x,n*sizeof(float),hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(y_d,y,n*sizeof(float),hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(z_d,z,n*sizeof(float),hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(u_d,u,n*sizeof(float),hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(v_d,v,n*sizeof(float),hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(I1_d,I1,1*sizeof(float),hipMemcpyHostToDevice) == hipSuccess);


    /* launch and time kernel code */
    hipEventRecord( start, 0 );  

    integrater<<<(n+NTPB-1)/NTPB,NTPB,NTPB*sizeof(float)>>>(x_d,y_d, z_d, u_d, v_d, I1_d,n);

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &times, start, stop );

    assert(hipMemcpy(I1,I1_d,1*sizeof(float),hipMemcpyDeviceToHost) == hipSuccess);
    *I1 = *I1/(float)n;
    printf("value: %f\nerror: %f\ntime elapsed: %f(s)\n", *I1, fabs(*I1-actual)/actual, times);
    hipFree(x_d);  hipFree(y_d);  hipFree(z_d);
    hipFree(I1_d);  hipFree(u_d);  hipFree(v_d);

    hipEventDestroy( start );
    hipEventDestroy( stop );


}

